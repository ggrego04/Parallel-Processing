#include "hip/hip_runtime.h"
/********************************************************
 * numberCountCUDA_Unified_solution.c.c
 * Login to machine 103ws??.in.cs.ucy.ac.cy
 * Compile Using:
 *  nvcc  -O3 numberCountCUDA_Unified_solution.cu
 *  ./a.out 100000099 100 3
 *  Number of Arguments 4
 * Array Size: 100000099 SetSize = 100 CountNumber 3
 * Initializing the array with Uniformly Distributed Data...
 * Counting 3 in Uniformly Distributed Data
 * Time: 0.10 Sec. SERIAL: Number if instances found 1000212 (1.000211%).
 * Kernel ONLY:Time: 0.04 Sec.
 * Time: 0.04 Sec. CUDA: Number if instances found 1000212 (1.000211%).
 */
#include <stdio.h>
#include "support.h"
// This Version is configured for GTX750Ti
// Consider one Block per SM
#define ID 999999
#define NUMBER_BLOCKS 10
#define NUMBER_THREADS_PER_BLOCK 32*32
// The size of the Information sent to threads
#define INFO_SIZE 4

void initData(char * vector, int size) {
	int i;
	srand (ID);
	for (i = 0; i < size; i++){
		vector[i] = rand()%26+'a';
		//vector[i] = 'a';
	}
}
void verify(unsigned int  * counters_CPU,unsigned int  * counters_AVX2){
	int j;
	for(j = 0; j < 26; ++j) 
		if (counters_CPU[j]!=counters_AVX2[j]) 
			printf("Does not Verify at value %d\n",j);

}

/* Count the instances of countNumber */
void countNumb_Serial(int size, unsigned int * counters, char * theArray, char from, char to){
  int i;
  for (i = 0; i < size; i++){
	 if (theArray[i]>=from && theArray[i]<=to)                          
	    counters[theArray[i]-'a']++;
  }
}

/***********************************
 * CUDA Implementation
 **********************************/
__global__ void numCountKernel(char *dArray, unsigned int *dCounters, unsigned int * dInfo){
	int i;
	int arraySize = dInfo[0];
	int totalNumberOfThreads =dInfo[1];
    char from = dInfo[2];
    char to = dInfo[3];
	int blockID = blockIdx.x;
	//int threadID = threadIdx.x;
	// Calculate the Unique Thread ID
	int threadUniqueID = threadIdx.x + blockIdx.x * blockDim.x;
    // Each WARP check contigues values in the global memory
	for (i=threadUniqueID;i<arraySize;i+=totalNumberOfThreads)
		if (dArray[i]>=from && dArray[i]<=to) 
            atomicAdd(&dCounters[dArray[i]-'a'+26*blockID],1);
}

void countNumber_CUDA(int arraySize, unsigned int * counters, char * theArray, char from, char to, int blocks, int threads){
	  int totalNumberOfThreads = blocks*threads;
	  int i=0;
	  unsigned int *Counters_CUDA;
      unsigned int *Info;
	  // We will need one Counter per BLOCK
	  int CountersInBytes = blocks*sizeof(unsigned int);
	   // Set the information send to threads
	   // dInfo: [0] the size of the dArray, [1] the countNumber, [1] the Total # of threads
	   // Allocate memory on the CPU for hInfo
	   // *** Using Unified Memory for CUDA ***
	   hipMallocManaged(&Info, sizeof(unsigned int)*INFO_SIZE);
	   Info[0] = arraySize;
	   Info[1] = totalNumberOfThreads;
       Info[2] = from;
       Info[3] = to;
	
	// Allocate Memory on the CPU for the Counters
	hipMallocManaged(&Counters_CUDA, sizeof(unsigned int)*CountersInBytes);
	
	// kernel invocation code
	dim3 dimBlock(threads);
	dim3 dimGrid(blocks);
	//Total Time: ?? Sec. Vs Kernel Only Time ?? Sec.
	//Computation ?? Sec Data Transfer ??Sec.
	startTime(1);
	numCountKernel<<<dimGrid, dimBlock>>>(theArray,Counters_CUDA,Info);
	//numCountKernel_shareVar<<<dimGrid, dimBlock>>>(dArray,dCounters,dInfo);
	hipDeviceSynchronize();
	stopTime(1);printf("Kernel ONLY:");elapsedTime(1);printf("\n");
	hipError_t  code = hipGetLastError();
	if (code != hipSuccess){ 
		printf("6) **%s** in %s at line %d\n", hipGetErrorString(code), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
	int j;
	// Reduction on Host
	for(i=0;i<blocks;i++){
		for(j=0;j<26;j++)
		counters[j] += Counters_CUDA[j+26*i];
	}
    // free the memory allocated on the GPU
    hipFree(Counters_CUDA);
	hipFree(Info);
	
}

int main(int argc, char **argv)
{
	int i;
    int blocks=NUMBER_BLOCKS;
    int threads=NUMBER_THREADS_PER_BLOCK;
	int  ArraySize=0, countNumber=0;
    char from, to;

	if (argc < 4){
		printf("Use: numberCount.out <ArraySize> <from> <to>\n");
		printf("NOTE: if numberOfChunks/chankSize > 100 the the number represents chankSize else is numberOfChunks\n");
	}else{
		ArraySize = atoi(argv[1]);
		from = argv[2][0];
		to = argv[3][0];
	}
	//printf("Number of Arguments %d\n",argc);
	//printf("Array Size: %d ArraySize = %d CountNumber %d\n", ArraySize, setSize,countNumber);
     __attribute__ ((aligned (256))) char * theArray = (char *) malloc(sizeof(char) * ArraySize);

    unsigned int  * counters_CPU = (unsigned int *) malloc(sizeof(unsigned int) * 26);
	unsigned int  * counters_CUDA = (unsigned int *) malloc(sizeof(unsigned int) * 26); 
    
	// *** Using Unified Memory for CUDA ***
	hipMallocManaged(&theArray, sizeof(int)*ArraySize);
  /**********************************************************************/
    initData(theArray, ArraySize);
  /**********************************************************************/
	#ifdef DEBUG
		printf("Uniformly Distributed Data:\n");
		printArray(theArray,ArraySize);
	#endif
	printf("Counting %d in Uniformly Distributed Data\n",countNumber);
    
	startTime(0);
	countNumb_Serial(ArraySize, counters_CPU, theArray, from, to);
	stopTime(0);for (i=0; i<26; i++) printf("%c Found %d times.\n", 'a'+i,counters_CPU[i]);
  	printf("Time CPU:"); elapsedTime(0);
    
	startTime(0);
	countNumber_CUDA(ArraySize, counters_CUDA,theArray, from, to, blocks, threads);
	stopTime(0);for (i=0; i<26; i++) printf("%c Found %d times.\n", 'a'+i,counters_CUDA[i]);
  	printf("Time CUDA:"); elapsedTime(0);

	verify(counters_CPU, counters_CUDA);
  /**********************************************************************/
   hipFree(theArray);
  return 0;
} 	

